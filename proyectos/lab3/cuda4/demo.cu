#include "hip/hip_runtime.h"
/*
  ======================================================================
   demo.c --- protoype to show off the simple solver
  ----------------------------------------------------------------------
   Author : Jos Stam (jstam@aw.sgi.com)
   Creation Date : Jan 9 2003

   Description:

    This code is a simple prototype that demonstrates how to use the
    code provided in my GDC2003 paper entitles "Real-Time Fluid Dynamics
    for Games". This code uses OpenGL and GLUT for graphics and interface

  =======================================================================
*/

#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <GL/glut.h>

#include <thrust/device_vector.h> // API para la reduccion del maximo
#include <hip/hip_runtime.h>           // API de cuda
#include <cutil_inline.h>   // Funciones para chequeo de errores

#include "timing.h"

/* macros */

#define IX(i,j) ((i)+(N+2)*(j))
#define DIV_CEIL(n, m) ((n) + (m) -1) / (m)

#define OPTIMAL_BLOCK_WIDTH 32
#define OPTIMAL_BLOCK_HEIGHT 5

/* external definitions (from solver.c) */

extern "C" void dens_step ( int N, float * x, float * x0, float * u, float * v, float diff, float dt );
extern "C" void vel_step ( int N, float * u, float * v, float * u0, float * v0, float visc, float dt );

/* global variables */

static int N;
static float dt, diff, visc;
static float force, source;
static int dvel;

static float * u, * v, * u_prev, * v_prev;
static float * dens, * dens_prev;

static int win_id;
static int win_x, win_y;
static int mouse_down[3];
static int omx, omy, mx, my;

/* global helper variables */

static float * vel2; // calculo de u*u + v*v
static float * host_u, * host_v, * host_dens; // renderizado

int BLOCK_WIDTH, BLOCK_HEIGHT; // deben ser importado desde el solver

/*
  ----------------------------------------------------------------------
   free/clear/allocate simulation data
  ----------------------------------------------------------------------
*/


static void free_data ( void )
{
    // libero memoria en el device
	if ( u ) cutilSafeCall(hipFree( u ));
	if ( v ) cutilSafeCall(hipFree( v ));
	if ( u_prev ) cutilSafeCall(hipFree( u_prev ));
	if ( v_prev ) cutilSafeCall(hipFree( v_prev ));
	if ( dens ) cutilSafeCall(hipFree( dens ));
	if ( dens_prev ) cutilSafeCall(hipFree( dens_prev ));
    	
	if ( vel2 ) cutilSafeCall(hipFree( vel2 ));    
    
    // libero memoria en el host
	if ( host_u ) free( host_u );
	if ( host_v ) free( host_v );
	if ( host_dens ) free( host_dens );
}

static void clear_data ( void )
{
	int size_bytes = (N+2) * (N+2) * sizeof(float);

    // seteo la memoria en el device a 0
    cutilSafeCall(hipMemset(u, 0, size_bytes));
    cutilSafeCall(hipMemset(dens, 0, size_bytes));
    cutilSafeCall(hipMemset(v, 0, size_bytes));
    cutilSafeCall(hipMemset(u_prev, 0, size_bytes));
    cutilSafeCall(hipMemset(v_prev, 0, size_bytes));
    cutilSafeCall(hipMemset(dens_prev, 0, size_bytes));

    cutilSafeCall(hipMemset(vel2, 0, size_bytes));
}

static int allocate_data ( void )
{
	int size_bytes = (N+2) * (N+2) * sizeof(float);
	
    // reservo memoria en el device
    cutilSafeCall(hipMalloc(&u, size_bytes));
    cutilSafeCall(hipMalloc(&v, size_bytes));
    cutilSafeCall(hipMalloc(&dens, size_bytes));
    cutilSafeCall(hipMalloc(&u_prev, size_bytes));
    cutilSafeCall(hipMalloc(&v_prev, size_bytes));
    cutilSafeCall(hipMalloc(&dens_prev, size_bytes));

    cutilSafeCall(hipMalloc(&vel2, size_bytes));
	
	//reservo memoria en el host
	host_u		= (float *) malloc ( size_bytes );
	host_v		= (float *) malloc ( size_bytes );
    host_dens   = (float *) malloc ( size_bytes );
    
    if (!host_u || !host_v || !host_dens) {
		fprintf ( stderr, "cannot allocate data\n" );
		return ( 0 );
	}

	return ( 1 );
}


/*
  ----------------------------------------------------------------------
   OpenGL specific drawing routines
  ----------------------------------------------------------------------
*/

static void pre_display ( void )
{
	glViewport ( 0, 0, win_x, win_y );
	glMatrixMode ( GL_PROJECTION );
	glLoadIdentity ();
	gluOrtho2D ( 0.0, 1.0, 0.0, 1.0 );
	glClearColor ( 0.0f, 0.0f, 0.0f, 1.0f );
	glClear ( GL_COLOR_BUFFER_BIT );
}

static void post_display ( void )
{
	glutSwapBuffers ();
}

static void draw_velocity ( void )
{
	// copio memoria auxiliar del device al host para el renderizado
    int size_bytes = (N+2) * (N+2) * sizeof(float);
	cutilSafeCall(hipMemcpy(host_u, u, size_bytes, hipMemcpyDefault));
	cutilSafeCall(hipMemcpy(host_v, v, size_bytes, hipMemcpyDefault));

	int i, j;
	float x, y, h;

	h = 1.0f/N;

	glColor3f ( 1.0f, 1.0f, 1.0f );
	glLineWidth ( 1.0f );

	glBegin ( GL_LINES );

		for ( i=1 ; i<=N ; i++ ) {
			x = (i-0.5f)*h;
			for ( j=1 ; j<=N ; j++ ) {
				y = (j-0.5f)*h;

				glVertex2f ( x, y );
				glVertex2f ( x + host_u[IX(i,j)], y + host_v[IX(i,j)] );
			}
		}

	glEnd ();
}

static void draw_density ( void )
{
	// copio memoria auxiliar del device al host para el renderizado
    int size_bytes = (N+2) * (N+2) * sizeof(float);
	cutilSafeCall(hipMemcpy(host_dens, dens, size_bytes, hipMemcpyDefault));

	int i, j;
	float x, y, h, d00, d01, d10, d11;

	h = 1.0f/N;

	glBegin ( GL_QUADS );

		for ( i=0 ; i<=N ; i++ ) {
			x = (i-0.5f)*h;
			for ( j=0 ; j<=N ; j++ ) {
				y = (j-0.5f)*h;

				d00 = host_dens[IX(i,j)];
				d01 = host_dens[IX(i,j+1)];
				d10 = host_dens[IX(i+1,j)];
				d11 = host_dens[IX(i+1,j+1)];

				glColor3f ( d00, d00, d00 ); glVertex2f ( x, y );
				glColor3f ( d10, d10, d10 ); glVertex2f ( x+h, y );
				glColor3f ( d11, d11, d11 ); glVertex2f ( x+h, y+h );
				glColor3f ( d01, d01, d01 ); glVertex2f ( x, y+h );
			}
		}

	glEnd ();
}

/*
  ----------------------------------------------------------------------
   relates mouse movements to forces sources
  ----------------------------------------------------------------------
*/

__global__ static void velocity2(unsigned int n, const float* u, const float* v, float* vel2){

    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    uint idx = y * n + x;
    if (x < n && y < n){
        vel2[idx] = u[idx] * u[idx] + v[idx] * v[idx];
    }
}

__global__ static void init_u_v_d(unsigned int n, float * d, float * u, float * v, 
                                  float max_velocity2, float max_density, 
                                  float force, float source, 
                                  int mouse_down0, int mouse_down2, 
                                  int mx, int my, int omx, int omy,
                                  int win_x, int win_y) {
    // un solo hilo inicializa u, v, d
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x == 0){
	    uint idx = (n / 2) * (n + 2) +  n / 2;
	    if (max_velocity2<0.0000005f) {
		    u[idx] = force * 10.0f;

		    v[idx] = force * 10.0f;
	    }
	    if (max_density<1.0f) {
		    d[idx] = source * 10.0f;
	    }
    
        if ( mouse_down0 || mouse_down2 ) return;

	    int i = (int)((       mx /(float)win_x)*n+1);
	    int j = (int)(((win_y-my)/(float)win_y)*n+1);

	    if ( i<1 || i>n || j<1 || j>n ) return;

	    if ( mouse_down0 ) {
		    u[j * (n+2) + i] = force * (mx-omx);
		    v[j * (n+2) + i] = force * (omy-my);
	    }

	    if ( mouse_down2 ) {
		    d[j * (n+2) + i] = source;
	    }
    }
}

void update_omx_omy() {
    // modifico omx, omy en el host
    if ( mouse_down[0] || mouse_down[2] ) return;
    int i = (int)((       mx /(float)win_x)*N+1);
    int j = (int)(((win_y-my)/(float)win_y)*N+1);
	if ( i<1 || i>N || j<1 || j>N ) return;
    omx = mx;
    omy = my;
}

static void react ( float * d, float * u, float * v )
{
	int size_bytes = (N+2) * (N+2) * sizeof(float);	    
    dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
    dim3 grid(DIV_CEIL(N+2, block.x), DIV_CEIL(N+2, block.y));

    // calculo velocidad    
    velocity2<<<grid, block>>>(N+2, u, v, vel2);
    CUT_CHECK_ERROR("Error al calcular u*u + v*v :");
    cutilSafeCall(hipDeviceSynchronize()); // espero a que termine el kernel   

    // calculo maxima velocidad
    thrust::device_ptr<float> thrust_vel2(vel2);
    float max_velocity2 = *thrust::max_element(thrust_vel2, thrust_vel2 + (N+2)*(N+2));
        
    // calculo maxima densidad
    thrust::device_ptr<float> thrust_d(d);
    float max_density = *thrust::max_element(thrust_d, thrust_d + (N+2)*(N+2));

    // seteo u, v, d a 0
    cutilSafeCall(hipMemset(u, 0, size_bytes));
    cutilSafeCall(hipMemset(v, 0, size_bytes));
    cutilSafeCall(hipMemset(d, 0, size_bytes));
    
    // inicializo u, v, d
    init_u_v_d<<<dim3(1), dim3(BLOCK_WIDTH)>>>(N, d, u, v, max_velocity2, max_density, 
                                               force, source, mouse_down[0], mouse_down[2], 
                                               mx, my, omx, omy, win_x, win_y);
    CUT_CHECK_ERROR("Error al inicializar u, v, d :");
    cutilSafeCall(hipDeviceSynchronize()); // espero a que termine el kernel
    update_omx_omy();
}

/*
  ----------------------------------------------------------------------
   GLUT callback routines
  ----------------------------------------------------------------------
*/

static void key_func ( unsigned char key, int x, int y )
{
	switch ( key )
	{
		case 'c':
		case 'C':
			clear_data ();
			break;

		case 'q':
		case 'Q':
			free_data ();
			exit ( 0 );
			break;

		case 'v':
		case 'V':
			dvel = !dvel;
			break;
	}
}

static void mouse_func ( int button, int state, int x, int y )
{
	omx = mx = x;
	omx = my = y;

	mouse_down[button] = state == GLUT_DOWN;
}

static void motion_func ( int x, int y )
{
	mx = x;
	my = y;
}

static void reshape_func ( int width, int height )
{
	glutSetWindow ( win_id );
	glutReshapeWindow ( width, height );

	win_x = width;
	win_y = height;
}

static void idle_func ( void )
{
	static int times = 1;
	static double start_t = 0.0;
	static double one_second = 0.0;
	static double react_ns_p_cell = 0.0;
	static double vel_ns_p_cell = 0.0;
	static double dens_ns_p_cell = 0.0;

	start_t = wtime();
	react ( dens_prev, u_prev, v_prev );
	react_ns_p_cell += 1.0e9 * (wtime()-start_t)/(N*N);

	start_t = wtime();
	vel_step ( N, u, v, u_prev, v_prev, visc, dt );
	vel_ns_p_cell += 1.0e9 * (wtime()-start_t)/(N*N);

	start_t = wtime();
	dens_step ( N, dens, dens_prev, u, v, diff, dt );
	dens_ns_p_cell += 1.0e9 * (wtime()-start_t)/(N*N);

	if (1.0<wtime()-one_second) { /* at least 1s between stats */
		printf("%lf, %lf, %lf, %lf: ns per cell total, react, vel_step, dens_step\n",
			(react_ns_p_cell+vel_ns_p_cell+dens_ns_p_cell)/times,
			react_ns_p_cell/times, vel_ns_p_cell/times, dens_ns_p_cell/times);
		one_second = wtime();
		react_ns_p_cell = 0.0;
		vel_ns_p_cell = 0.0;
		dens_ns_p_cell = 0.0;
		times = 1;
	} else {
		times++;
	}

	glutSetWindow ( win_id );
	glutPostRedisplay ();
}

static void display_func ( void )
{
	pre_display ();

		if ( dvel ) draw_velocity ();
		else		draw_density ();

	post_display ();
}


/*
  ----------------------------------------------------------------------
   open_glut_window --- open a glut compatible window and set callbacks
  ----------------------------------------------------------------------
*/

static void open_glut_window ( void )
{
	glutInitDisplayMode ( GLUT_RGBA | GLUT_DOUBLE );

	glutInitWindowPosition ( 0, 0 );
	glutInitWindowSize ( win_x, win_y );
	win_id = glutCreateWindow ( "Alias | wavefront" );

	glClearColor ( 0.0f, 0.0f, 0.0f, 1.0f );
	glClear ( GL_COLOR_BUFFER_BIT );
	glutSwapBuffers ();
	glClear ( GL_COLOR_BUFFER_BIT );
	glutSwapBuffers ();

	pre_display ();

	glutKeyboardFunc ( key_func );
	glutMouseFunc ( mouse_func );
	glutMotionFunc ( motion_func );
	glutReshapeFunc ( reshape_func );
	glutIdleFunc ( idle_func );
	glutDisplayFunc ( display_func );
}


/*
  ----------------------------------------------------------------------
   main --- main routine
  ----------------------------------------------------------------------
*/

int main ( int argc, char ** argv )
{
	glutInit ( &argc, argv );

	if ( argc != 1 && argc != 4 && argc != 6 ) {
		fprintf ( stderr, "usage : %s N dt diff visc force source\n", argv[0] );
		fprintf ( stderr, "where:\n" );\
		fprintf ( stderr, "\t N      : grid resolution\n" );
		fprintf ( stderr, "\t dt     : time step\n" );
		fprintf ( stderr, "\t diff   : diffusion rate of the density\n" );
		fprintf ( stderr, "\t visc   : viscosity of the fluid\n" );
		fprintf ( stderr, "\t force  : scales the mouse movement that generate a force\n" );
		fprintf ( stderr, "\t source : amount of density that will be deposited\n" );
		exit ( 1 );
	}

	if ( argc == 1 ) {
		N = 128;
		dt = 0.1f;
		diff = 0.0f;
		visc = 0.0f;
		force = 5.0f;
		source = 100.0f;
		fprintf ( stderr, "Using defaults : N=%d dt=%g diff=%g visc=%g force = %g source=%g\n",
			N, dt, diff, visc, force, source );
	    BLOCK_WIDTH = OPTIMAL_BLOCK_WIDTH;
	    BLOCK_HEIGHT = OPTIMAL_BLOCK_HEIGHT;
	} else if (argc == 4) {
	    N = atoi( argv[1] );
	    dt = 0.1f;
		diff = 0.0f;
		visc = 0.0f;
		force = 5.0f;
		source = 100.0f;
		fprintf ( stderr, "Using defaults : N=%d dt=%g diff=%g visc=%g force = %g source=%g\n",
			N, dt, diff, visc, force, source );		
	    BLOCK_WIDTH = atoi( argv [2] );
	    BLOCK_HEIGHT = atoi( argv [3] );
	} else {
		N = atoi(argv[1]);
		dt = atof(argv[2]);
		diff = atof(argv[3]);
		visc = atof(argv[4]);
		force = atof(argv[5]);
		source = atof(argv[6]);
	    BLOCK_WIDTH = OPTIMAL_BLOCK_WIDTH;
	    BLOCK_HEIGHT = OPTIMAL_BLOCK_HEIGHT;
	}

	assert (N > 0 && BLOCK_WIDTH > 0 && BLOCK_HEIGHT > 0);

	dvel = 0;

	if ( !allocate_data () ) exit ( 1 );
	clear_data ();

	win_x = 512;
	win_y = 512;
	open_glut_window ();

	glutMainLoop ();

	exit ( 0 );
}
